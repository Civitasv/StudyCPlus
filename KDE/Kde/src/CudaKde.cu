#include "hip/hip_runtime.h"
#include <>
#include <thrust/copy.h>
#include <thrust/device_malloc.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include <chrono>
#include <memory>

#include "CudaKde.cuh"
#include "Point.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>

#include "Renderer.h"
#include "glm/glm.hpp"
#include "glm/gtx/transform.hpp"

namespace kde {
__device__ float GPUDist(const Point& p1, const Point& p2) {
  double a = p1.lon - p2.lon;
  double b = p1.lat - p2.lat;

  return a * a + b * b;
}

__device__ float GPUKernel(float t) {
  float val = (1 - t * t);
  return (3 / M_PI) * val * val;
}

__global__ void CalCulateKDEKernel(const Point* pts, const Point* calculate_pts,
                                   Item* res, float band_width, int width,
                                   int height, int pts_size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index > width * height) return;
  Point p = calculate_pts[index];

  float f_estimate = 0.0f;

  for (int m = 0; m < pts_size; m++) {
    float distance = GPUDist(pts[m], p);
    /*printf("index: %d, x: %f,y: %f, px: %f, py: %f ---- DISTANCE: %f \n",
       index, m, pts[m].lon, pts[m].lat, p.lon, p.lat, distance);*/

    if (distance < band_width * band_width) {
      f_estimate += GPUKernel(distance / band_width);
    }
  }

  /*printf(
      "index: %d, pts_size: %d, band width: %f, width: %d, height: %d, "
      "estimate: %f, min: %f, max: %f\n",
      index, pts_size, band_width, f_estimate);*/

  f_estimate = f_estimate / (pts_size * band_width * band_width);
  res[index].value = f_estimate;
  res[index].col = index % width;
  res[index].row = index / width;
}

__global__ void CalCulateOpenGLVBO(Item* res, float* data, int width,
                                   int height) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // printf("CalCulateOpenGLVBO: index: %d", index);
  if (index > width * height) return;

  float min = res[0].value;
  float max = res[width * height - 1].value;

  float value = (res[index].value - min) / (max - min);
  int d_index = index * 5;
  int row = res[index].row;
  int col = res[index].col;

  data[d_index + 0] = col;
  data[d_index + 1] = row;
  if (value == 0) {
    data[d_index + 2] = 1;
    data[d_index + 3] = 1;
    data[d_index + 4] = 1;
  } else if (value > 0 && value < 0.4) {
    data[d_index + 2] = 1;
    data[d_index + 3] = 1;
    data[d_index + 4] = 0;
  } else if (value >= 0.4 && value < 0.7) {
    data[d_index + 2] = 0;
    data[d_index + 3] = 1;
    data[d_index + 4] = 0;
  } else if (value >= 0.7 && value < 0.9) {
    data[d_index + 2] = 0;
    data[d_index + 3] = 0;
    data[d_index + 4] = 1;
  } else if (value >= 0.9 && value <= 1) {
    data[d_index + 2] = 1;
    data[d_index + 3] = 0;
    data[d_index + 4] = 0;
  }
}

void PrintCUDAInfo() {
  int dev = 0;
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, dev);
  std::cout << "ʹ��GPU device " << dev << ": " << devProp.name << std::endl;
  std::cout << "SM��������" << devProp.multiProcessorCount << std::endl;
  std::cout << "ÿ���߳̿�Ĺ����ڴ��С��"
            << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
  std::cout << "ÿ���߳̿������߳�����" << devProp.maxThreadsPerBlock
            << std::endl;
  std::cout << "ÿ��SM������߳�����" << devProp.maxThreadsPerMultiProcessor
            << std::endl;
  std::cout << "ÿ��SM������߳�������"
            << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
}

struct sort_functor {
  __host__ __device__ bool operator()(Item left, Item right) const {
    return (left.value < right.value);
  }
};

RendererElement GPUKde(std::vector<Point>& pts, Rect& rect, int width,
                       int height) {
  using namespace std::chrono;
  auto start = high_resolution_clock::now();

  PrintCUDAInfo();

  thrust::host_vector<Point> h_calculation_pts(width * height);
  thrust::device_vector<Point> d_pts(pts);
  thrust::device_vector<Item> d_res(width * height);

  // Calculate band width
  Point avePt = ave(pts);
  float band_width = h(pts, avePt);
  rect.top += band_width;
  rect.bottom -= band_width;
  rect.left -= band_width;
  rect.right += band_width;

  float item_w = (rect.right - rect.left) / width;
  float item_h = (rect.top - rect.bottom) / height;

  // Calculate kernel density estimation
  for (int x = 0; x < width; x++) {
    float item_x = rect.left + item_w * x;

    for (int y = 0; y < height; y++) {
      float item_y = rect.bottom + item_h * y;
      h_calculation_pts[y * width + x] = {item_x, item_y};
    }
  }
  thrust::device_vector<Point> d_calculation_pts(h_calculation_pts);

  dim3 threadsPerBlock(64);
  dim3 blockPerGrid((width * height + threadsPerBlock.x - 1) /
                    threadsPerBlock.x);

  CalCulateKDEKernel<<<blockPerGrid, threadsPerBlock>>>(
      raw_pointer_cast(&d_pts[0]), raw_pointer_cast(&d_calculation_pts[0]),
      raw_pointer_cast(&d_res[0]), band_width, width, height, pts.size());
  if (const hipError_t err = hipGetLastError()) {
    std::cout << "CUDA KDE KERNEL ERROR: " << hipGetErrorString(err) << '\n';
  }
  thrust::stable_sort(d_res.begin(), d_res.end(), sort_functor());

  size_t size = width * height;
  VertexArray* va = new VertexArray();
  VertexBuffer* vb = new VertexBuffer(width * height * 5 * sizeof(float), size);

  VertexBufferLayout layout;
  layout.Push<float>(2);
  layout.Push<float>(3);
  // index buffer object, specify how I want to draw this layout.

  float* data;
  hipGraphicsResourceGetMappedPointer((void**)&data, &size, vb->vbo_cuda);

  CalCulateOpenGLVBO<<<blockPerGrid, threadsPerBlock>>>(
      raw_pointer_cast(&d_res[0]), data, width, height);
  if (const hipError_t err = hipGetLastError()) {
    std::cout << "CUDA OPENGL ERROR: " << hipGetErrorString(err) << '\n';
  }

  hipGraphicsUnmapResources(1, &vb->vbo_cuda, NULL);
  va->AddBuffer(*vb, layout);
  
  // 4 * 3
  glm::mat4 proj =
      glm::ortho(0.0f, width * 1.0f, 0.0f, height * 1.0f, -1.0f, 1.0f);

  Shader* shader = new Shader("res/shaders/Basic.shader");
  shader->Bind();
  shader->SetUniformMat4f("u_MVP", proj);

  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<milliseconds>(stop - start);
  std::cout << "CALCULATION TIME:: " << duration.count() << " ms" << std::endl;

  return {shader, va, vb, NULL};
}

RendererElement GPUCalculate() {
  // 1. Read file
  auto data = Coordinates("res/data/coord.txt");
  auto pts = data.first;
  auto rect = data.second;
  // 2. Calculate kde
  int width = 15000;
  int height =
      floor(width * (rect.top - rect.bottom) / (rect.right - rect.left));

  RendererElement res = GPUKde(pts, rect, width, height);

  // 3. Return and let renderer to plot
  return res;
}
}  // namespace kde