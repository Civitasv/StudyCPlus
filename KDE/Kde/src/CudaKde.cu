#include "hip/hip_runtime.h"
#include <>
#include <thrust/copy.h>
#include <thrust/device_malloc.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include <chrono>
#include <memory>

#include "CudaKde.cuh"
#include "Point.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>

#include "Renderer.h"
#include "glm/glm.hpp"
#include "glm/gtx/transform.hpp"

namespace kde {
struct RGB {
  uint8_t r;
  uint8_t g;
  uint8_t b;
};

__device__ float GPUDist(const Point& p1, const Point& p2) {
  double a = p1.lon - p2.lon;
  double b = p1.lat - p2.lat;

  return a * a + b * b;
}

__device__ float GPUKernel(float t) {
  float val = (1 - t * t);
  return (3 / M_PI) * val * val;
}

__device__ RGB rgb(double ratio) {
  // we want to normalize ratio so that it fits in to 6 regions
  // where each region is 256 units long
  int val = 256;
  int normalized = int(ratio * val);

  // find the region for this position
  int region = normalized / val;

  // find the distance to the start of the closest region
  int x = normalized % val;

  uint8_t r = 0, g = 0, b = 0;
  switch (region) {
    case 0:
      r = val - 1;
      g = val - 1;
      b = val - 1;
      r -= x;
      g -= x;
      b -= x;
      break;
    case 1:
      r = 0;
      g = 0;
      b = 255;
      r += x;
      break;
  }
  return {r, g, b};
}

__global__ void CalCulateKDEKernel(const Point* pts, const Point* calculate_pts,
                                   Item* res, float band_width, int width,
                                   int height, int pts_size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index > width * height) return;
  Point p = calculate_pts[index];

  float f_estimate = 0.0f;

  for (int m = 0; m < pts_size; m++) {
    float distance = GPUDist(pts[m], p);
    /*printf("index: %d, x: %f,y: %f, px: %f, py: %f ---- DISTANCE: %f \n",
       index, m, pts[m].lon, pts[m].lat, p.lon, p.lat, distance);*/

    if (distance < band_width) {
      f_estimate += GPUKernel(distance / band_width);
    }
  }

  /*printf(
      "index: %d, pts_size: %d, band width: %f, width: %d, height: %d, "
      "estimate: %f, min: %f, max: %f\n",
      index, pts_size, band_width, f_estimate);*/

  f_estimate = f_estimate / (pts_size * band_width * band_width);
  res[index].value = f_estimate;
  res[index].col = index % width;
  res[index].row = index / width;
}

__global__ void CalCulateOpenGLVBO(Item* res, float* data, int width,
                                   int height, int max_pos, int min_pos) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // printf("CalCulateOpenGLVBO: index: %d", index);
  if (index > width * height) return;

  float min = res[min_pos].value;
  float max = res[max_pos].value;

  float val = (res[index].value - min) / (max - min);
  // float value = res[index].value;
  int count = index * 5;
  int row = res[index].row;
  int col = res[index].col;

  data[count + 0] = col;
  data[count + 1] = row;

  if (val == 0) {
    data[count + 2] = 255.0f / 255.0f;
    data[count + 3] = 255.0f / 255.0f;
    data[count + 4] = 255.0f / 255.0f;
  } else if (val > 0 && val < 0.1) {
    data[count + 2] = 170.0f / 255.0f;
    data[count + 3] = 170.0f / 255.0f;
    data[count + 4] = 170.0f / 255.0f;
  } else if (val >= 0.1 && val < 0.7) {
    data[count + 2] = 61.0f / 255.0f;
    data[count + 3] = 61.0f / 255.0f;
    data[count + 4] = 61.0f / 255.0f;
  } else if (val >= 0.7 && val < 0.9) {
    data[count + 2] = 33.0f / 255.0f;
    data[count + 3] = 33.0f / 255.0f;
    data[count + 4] = 33.0f / 255.0f;
  } else if (val >= 0.9 && val <= 1) {
    data[count + 2] = 24.0f / 255.0f;
    data[count + 3] = 24.0f / 255.0f;
    data[count + 4] = 24.0f / 255.0f;
  }
}

void PrintCUDAInfo() {
  int dev = 0;
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, dev);
  std::cout << "ʹ��GPU device " << dev << ": " << devProp.name << std::endl;
  std::cout << "SM��������" << devProp.multiProcessorCount << std::endl;
  std::cout << "ÿ���߳̿�Ĺ����ڴ��С��"
            << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
  std::cout << "ÿ���߳̿������߳�����" << devProp.maxThreadsPerBlock
            << std::endl;
  std::cout << "ÿ��SM������߳�����" << devProp.maxThreadsPerMultiProcessor
            << std::endl;
  std::cout << "ÿ��SM������߳�������"
            << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
}

struct sort_functor {
  __host__ __device__ bool operator()(Item left, Item right) const {
    return (left.value < right.value);
  }
};

RendererElement GPUKde(std::vector<Point>& pts, Rect& rect, int width,
                       int height) {
  using namespace std::chrono;
  // auto start = high_resolution_clock::now();

  PrintCUDAInfo();

  thrust::host_vector<Point> h_calculation_pts(width * height);
  thrust::device_vector<Point> d_pts(pts);
  thrust::device_vector<Item> d_res(width * height);

  // Calculate band width
  Point avePt = ave(pts);
  float band_width = h(pts, avePt);
  rect.top += band_width;
  rect.bottom -= band_width;
  rect.left -= band_width;
  rect.right += band_width;

  float item_w = (rect.right - rect.left) / width;
  float item_h = (rect.top - rect.bottom) / height;

  // Calculate kernel density estimation
  for (int x = 0; x < width; x++) {
    float item_x = rect.left + item_w * x;

    for (int y = 0; y < height; y++) {
      float item_y = rect.bottom + item_h * y;
      h_calculation_pts[y * width + x] = {item_x, item_y};
    }
  }
  thrust::device_vector<Point> d_calculation_pts(h_calculation_pts);

  dim3 threadsPerBlock(256);
  dim3 blockPerGrid((width * height + threadsPerBlock.x - 1) /
                    threadsPerBlock.x);

  CalCulateKDEKernel<<<blockPerGrid, threadsPerBlock>>>(
      raw_pointer_cast(&d_pts[0]), raw_pointer_cast(&d_calculation_pts[0]),
      raw_pointer_cast(&d_res[0]), band_width, width, height, pts.size());
  if (const hipError_t err = hipGetLastError()) {
    std::cout << "CUDA KDE KERNEL ERROR: " << hipGetErrorString(err) << '\n';
  }
  /*auto stop = high_resolution_clock::now();
  auto duration = duration_cast<milliseconds>(stop - start);
  std::cout << "CALCULATION TIME:: " << duration.count() << " ms" <<
  std::endl;*/

  // start = high_resolution_clock::now();

  thrust::device_vector<Item>::iterator iter =
      thrust::max_element(d_res.begin(), d_res.end(), sort_functor());
  unsigned int max_position = iter - d_res.begin();

  thrust::device_vector<Item>::iterator iter2 =
      thrust::min_element(d_res.begin(), d_res.end(), sort_functor());
  unsigned int min_position = iter2 - d_res.begin();

  size_t size = width * height;
  VertexArray* va = new VertexArray();
  VertexBuffer* vb = new VertexBuffer(width * height * 5 * sizeof(float), size);

  VertexBufferLayout layout;
  layout.Push<float>(2);
  layout.Push<float>(3);
  // index buffer object, specify how I want to draw this layout.

  float* data;
  hipGraphicsResourceGetMappedPointer((void**)&data, &size, vb->vbo_cuda);

  CalCulateOpenGLVBO<<<blockPerGrid, threadsPerBlock>>>(
      raw_pointer_cast(&d_res[0]), data, width, height, max_position,
      min_position);
  if (const hipError_t err = hipGetLastError()) {
    std::cout << "CUDA OPENGL ERROR: " << hipGetErrorString(err) << '\n';
  }

  hipGraphicsUnmapResources(1, &vb->vbo_cuda, NULL);
  va->AddBuffer(*vb, layout);

  // 4 * 3
  glm::mat4 proj =
      glm::ortho(0.0f, width * 1.0f, 0.0f, height * 1.0f, -1.0f, 1.0f);

  Shader* shader = new Shader("res/shaders/Basic.shader");
  shader->Bind();
  shader->SetUniformMat4f("u_MVP", proj);

  // auto stop = high_resolution_clock::now();
  // auto duration = duration_cast<milliseconds>(stop - start);
  // std::cout << "PREPARE TIME:: " << duration.count() << " ms" << std::endl;

  return {shader, va, vb, NULL};
}

RendererElement GPUCalculate() {
  // 1. Read file
  auto data = Coordinates("res/data/coord.txt");
  auto pts = data.first;
  auto rect = data.second;
  float w = rect.right - rect.left;
  float h = rect.top - rect.bottom;

  if (w > h) {
    rect.top += (w - h) / 2;
    rect.bottom -= (w - h) / 2;
  } else {
    rect.right += (h - w) / 2;
    rect.left -= (h - w) / 2;
  }
  // 2. Calculate kde
  int width = 5000;
  int height = 5000;

  RendererElement res = GPUKde(pts, rect, width, height);

  // 3. Return and let renderer to plot
  return res;
}
}  // namespace kde